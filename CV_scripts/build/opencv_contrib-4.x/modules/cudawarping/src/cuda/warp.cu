#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2009, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#if !defined CUDA_DISABLER

#include "opencv2/core/cuda/common.hpp"
#include "opencv2/core/cuda/border_interpolate.hpp"
#include "opencv2/core/cuda/vec_traits.hpp"
#include "opencv2/core/cuda/vec_math.hpp"
#include "opencv2/core/cuda/saturate_cast.hpp"
#include "opencv2/core/cuda/filters.hpp"

namespace cv { namespace cuda { namespace device
{
    namespace imgproc
    {
        struct AffineTransform
        {
            static const int rows = 2;
            static __device__ __forceinline__ float2 calcCoord(const float warpMat[AffineTransform::rows * 3], int x, int y)
            {
                const float xcoo = warpMat[0] * x + warpMat[1] * y + warpMat[2];
                const float ycoo = warpMat[3] * x + warpMat[4] * y + warpMat[5];

                return make_float2(xcoo, ycoo);
            }

            struct Coefficients
            {
                Coefficients(const float* c_)
                {
                    for(int i = 0; i < AffineTransform::rows * 3; i++)
                        c[i] = c_[i];
                }
                float c[AffineTransform::rows * 3];
            };
        };

        struct PerspectiveTransform
        {
            static const int rows = 3;
            static __device__ __forceinline__ float2 calcCoord(const float warpMat[PerspectiveTransform::rows * 3], int x, int y)
            {
                const float coeff = 1.0f / (warpMat[6] * x + warpMat[7] * y + warpMat[8]);

                const float xcoo = coeff * (warpMat[0] * x + warpMat[1] * y + warpMat[2]);
                const float ycoo = coeff * (warpMat[3] * x + warpMat[4] * y + warpMat[5]);

                return make_float2(xcoo, ycoo);
            }
            struct Coefficients
            {
                Coefficients(const float* c_)
                {
                    for(int i = 0; i < PerspectiveTransform::rows * 3; i++)
                        c[i] = c_[i];
                }

                float c[PerspectiveTransform::rows * 3];
            };
        };

        ///////////////////////////////////////////////////////////////////
        // Build Maps

        template <class Transform> __global__ void buildWarpMaps(PtrStepSzf xmap, PtrStepf ymap, const typename Transform::Coefficients warpMat)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < xmap.cols && y < xmap.rows)
            {
                const float2 coord = Transform::calcCoord(warpMat.c, x, y);

                xmap(y, x) = coord.x;
                ymap(y, x) = coord.y;
            }
        }

        template <class Transform> void buildWarpMaps_caller(PtrStepSzf xmap, PtrStepSzf ymap, const float warpMat[Transform::rows * 3], hipStream_t stream)
        {
            dim3 block(32, 8);
            dim3 grid(divUp(xmap.cols, block.x), divUp(xmap.rows, block.y));

            buildWarpMaps<Transform><<<grid, block, 0, stream>>>(xmap, ymap, warpMat);
            cudaSafeCall( hipGetLastError() );

            if (stream == 0)
                cudaSafeCall( hipDeviceSynchronize() );
        }

        void buildWarpAffineMaps_gpu(float coeffs[2 * 3], PtrStepSzf xmap, PtrStepSzf ymap, hipStream_t stream)
        {
            buildWarpMaps_caller<AffineTransform>(xmap, ymap, coeffs, stream);
        }

        void buildWarpPerspectiveMaps_gpu(float coeffs[3 * 3], PtrStepSzf xmap, PtrStepSzf ymap, hipStream_t stream)
        {
            buildWarpMaps_caller<PerspectiveTransform>(xmap, ymap, coeffs, stream);
        }

        ///////////////////////////////////////////////////////////////////
        // Warp

        template <class Transform, class Ptr2D, typename T> __global__ void warp(const Ptr2D src, PtrStepSz<T> dst, const typename Transform::Coefficients warpMat)
        {
            const int x = blockDim.x * blockIdx.x + threadIdx.x;
            const int y = blockDim.y * blockIdx.y + threadIdx.y;

            if (x < dst.cols && y < dst.rows)
            {
                const float2 coord = Transform::calcCoord(warpMat.c, x, y);

                dst.ptr(y)[x] = saturate_cast<T>(src(coord.y, coord.x));
            }
        }

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcherStream
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], hipStream_t stream, bool)
            {
                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                BorderReader< PtrStep<T>, B<work_type> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, B<work_type> > > filter_src(brdSrc);

                warp<Transform><<<grid, block, 0, stream>>>(filter_src, dst, warpMat);
                cudaSafeCall( hipGetLastError() );
            }
        };

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcherNonStream
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], bool)
            {
                CV_UNUSED(xoff);
                CV_UNUSED(yoff);
                CV_UNUSED(srcWhole);

                typedef typename TypeVec<float, VecTraits<T>::cn>::vec_type work_type;

                dim3 block(32, 8);
                dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

                B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue));
                BorderReader< PtrStep<T>, B<work_type> > brdSrc(src, brd);
                Filter< BorderReader< PtrStep<T>, B<work_type> > > filter_src(brdSrc);

                warp<Transform><<<grid, block>>>(filter_src, dst, warpMat);
                cudaSafeCall( hipGetLastError() );

                cudaSafeCall( hipDeviceSynchronize() );
            }
        };

        #define OPENCV_CUDA_IMPLEMENT_WARP_TEX(type) \
            texture< type , hipTextureType2D > tex_warp_ ## type (0, hipFilterModePoint, hipAddressModeClamp); \
            struct tex_warp_ ## type ## _reader \
            { \
                typedef type elem_type; \
                typedef int index_type; \
                int xoff, yoff; \
                tex_warp_ ## type ## _reader (int xoff_, int yoff_) : xoff(xoff_), yoff(yoff_) {} \
                __device__ __forceinline__ elem_type operator ()(index_type y, index_type x) const \
                { \
                    return tex2D(tex_warp_ ## type , x + xoff, y + yoff); \
                } \
            }; \
            template <class Transform, template <typename> class Filter, template <typename> class B> struct WarpDispatcherNonStream<Transform, Filter, B, type> \
            { \
                static void call(PtrStepSz< type > src, PtrStepSz< type > srcWhole, int xoff, int yoff, PtrStepSz< type > dst, const float* borderValue, const float warpMat[Transform::rows*3], bool cc20) \
                { \
                    typedef typename TypeVec<float, VecTraits< type >::cn>::vec_type work_type; \
                    dim3 block(32, cc20 ? 8 : 4); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_warp_ ## type , srcWhole); \
                    tex_warp_ ## type ##_reader texSrc(xoff, yoff); \
                    B<work_type> brd(src.rows, src.cols, VecTraits<work_type>::make(borderValue)); \
                    BorderReader< tex_warp_ ## type ##_reader, B<work_type> > brdSrc(texSrc, brd); \
                    Filter< BorderReader< tex_warp_ ## type ##_reader, B<work_type> > > filter_src(brdSrc); \
                    warp<Transform><<<grid, block>>>(filter_src, dst, warpMat); \
                    cudaSafeCall( hipGetLastError() ); \
                    cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            }; \
            template <class Transform, template <typename> class Filter> struct WarpDispatcherNonStream<Transform, Filter, BrdReplicate, type> \
            { \
                static void call(PtrStepSz< type > src, PtrStepSz< type > srcWhole, int xoff, int yoff, PtrStepSz< type > dst, const float*, const float warpMat[Transform::rows*3], bool) \
                { \
                    dim3 block(32, 8); \
                    dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y)); \
                    bindTexture(&tex_warp_ ## type , srcWhole); \
                    tex_warp_ ## type ##_reader texSrc(xoff, yoff); \
                    if (srcWhole.cols == src.cols && srcWhole.rows == src.rows) \
                    { \
                        Filter< tex_warp_ ## type ##_reader > filter_src(texSrc); \
                        warp<Transform><<<grid, block>>>(filter_src, dst, warpMat); \
                    } \
                    else \
                    { \
                        BrdReplicate<type> brd(src.rows, src.cols); \
                        BorderReader< tex_warp_ ## type ##_reader, BrdReplicate<type> > brdSrc(texSrc, brd); \
                        Filter< BorderReader< tex_warp_ ## type ##_reader, BrdReplicate<type> > > filter_src(brdSrc); \
                        warp<Transform><<<grid, block>>>(filter_src, dst, warpMat); \
                    } \
                    cudaSafeCall( hipGetLastError() ); \
                    cudaSafeCall( hipDeviceSynchronize() ); \
                } \
            };

        OPENCV_CUDA_IMPLEMENT_WARP_TEX(uchar)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(uchar2)
        OPENCV_CUDA_IMPLEMENT_WARP_TEX(uchar4)

        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(schar)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(char2)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(char4)

        OPENCV_CUDA_IMPLEMENT_WARP_TEX(ushort)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(ushort2)
        OPENCV_CUDA_IMPLEMENT_WARP_TEX(ushort4)

        OPENCV_CUDA_IMPLEMENT_WARP_TEX(short)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(short2)
        OPENCV_CUDA_IMPLEMENT_WARP_TEX(short4)

        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(int)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(int2)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(int4)

        OPENCV_CUDA_IMPLEMENT_WARP_TEX(float)
        //OPENCV_CUDA_IMPLEMENT_WARP_TEX(float2)
        OPENCV_CUDA_IMPLEMENT_WARP_TEX(float4)

        #undef OPENCV_CUDA_IMPLEMENT_WARP_TEX

        template <class Transform, template <typename> class Filter, template <typename> class B, typename T> struct WarpDispatcher
        {
            static void call(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], hipStream_t stream, bool cc20)
            {
                if (stream == 0)
                    WarpDispatcherNonStream<Transform, Filter, B, T>::call(src, srcWhole, xoff, yoff, dst, borderValue, warpMat, cc20);
                else
                    WarpDispatcherStream<Transform, Filter, B, T>::call(src, dst, borderValue, warpMat, stream, cc20);
            }
        };

        template <class Transform, typename T>
        void warp_caller(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, PtrStepSzb dst, int interpolation,
                         int borderMode, const float* borderValue, const float warpMat[Transform::rows*3], hipStream_t stream, bool cc20)
        {
            typedef void (*func_t)(PtrStepSz<T> src, PtrStepSz<T> srcWhole, int xoff, int yoff, PtrStepSz<T> dst, const float* borderValue, const float warpMat[Transform::rows*3], hipStream_t stream, bool cc20);

            static const func_t funcs[3][5] =
            {
                {
                    WarpDispatcher<Transform, PointFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, PointFilter, BrdReflect101, T>::call
                },
                {
                    WarpDispatcher<Transform, LinearFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, LinearFilter, BrdReflect101, T>::call
                },
                {
                    WarpDispatcher<Transform, CubicFilter, BrdConstant, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReplicate, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReflect, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdWrap, T>::call,
                    WarpDispatcher<Transform, CubicFilter, BrdReflect101, T>::call
                }
            };

            funcs[interpolation][borderMode](static_cast< PtrStepSz<T> >(src), static_cast< PtrStepSz<T> >(srcWhole), xoff, yoff,
                                             static_cast< PtrStepSz<T> >(dst), borderValue, warpMat, stream, cc20);
        }

        template <typename T> void warpAffine_gpu(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation,
                                                  int borderMode, const float* borderValue, hipStream_t stream, bool cc20)
        {
            warp_caller<AffineTransform, T>(src, srcWhole, xoff, yoff, dst, interpolation, borderMode, borderValue, coeffs, stream, cc20);
        }

        template void warpAffine_gpu<uchar >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<uchar2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<uchar3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<uchar4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void warpAffine_gpu<schar>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<char2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<char3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<char4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<ushort >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<ushort2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<ushort3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<ushort4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<short >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<short2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<short3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<short4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void warpAffine_gpu<int >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<int2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<int3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<int4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpAffine_gpu<float >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpAffine_gpu<float2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<float3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpAffine_gpu<float4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[2 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template <typename T> void warpPerspective_gpu(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation,
                                                  int borderMode, const float* borderValue, hipStream_t stream, bool cc20)
        {
            warp_caller<PerspectiveTransform, T>(src, srcWhole, xoff, yoff, dst, interpolation, borderMode, borderValue, coeffs, stream, cc20);
        }

        template void warpPerspective_gpu<uchar >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<uchar2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<uchar3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<uchar4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void warpPerspective_gpu<schar>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<char2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<char3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<char4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<ushort >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<ushort2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<ushort3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<ushort4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<short >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<short2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<short3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<short4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        //template void warpPerspective_gpu<int >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<int2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<int3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<int4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);

        template void warpPerspective_gpu<float >(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        //template void warpPerspective_gpu<float2>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<float3>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
        template void warpPerspective_gpu<float4>(PtrStepSzb src, PtrStepSzb srcWhole, int xoff, int yoff, float coeffs[3 * 3], PtrStepSzb dst, int interpolation, int borderMode, const float* borderValue, hipStream_t stream, bool cc20);
    } // namespace imgproc
}}} // namespace cv { namespace cuda { namespace cudev


#endif /* CUDA_DISABLER */
